
#include <hip/hip_runtime.h>
// -----------------------------------------------------------------------------
// This file was autogenerated by symforce from template:
//     function/FUNCTION.cu.jinja
// Do NOT modify by hand.
// -----------------------------------------------------------------------------

namespace sym {

__host__ __device__ void BackendTestFunctionFloat32(
    const float x, const float y, float* const __restrict__ res0, float* const __restrict__ res1,
    float* const __restrict__ res2, float* const __restrict__ res3, float* const __restrict__ res4,
    float* const __restrict__ res5, float* const __restrict__ res6, float* const __restrict__ res7,
    float* const __restrict__ res8, float* const __restrict__ res9, float* const __restrict__ res10,
    float* const __restrict__ res11, float* const __restrict__ res12,
    float* const __restrict__ res13, float* const __restrict__ res14,
    float* const __restrict__ res15, float* const __restrict__ res16,
    float* const __restrict__ res17, float* const __restrict__ res18,
    float* const __restrict__ res19, float* const __restrict__ res20,
    float* const __restrict__ res21, float* const __restrict__ res22,
    float* const __restrict__ res23, float* const __restrict__ res24,
    float* const __restrict__ res25, float* const __restrict__ res26,
    float* const __restrict__ res27, float* const __restrict__ res28,
    float* const __restrict__ res29, float* const __restrict__ res30,
    float* const __restrict__ res31, float* const __restrict__ res32,
    float* const __restrict__ res33, float* const __restrict__ res34,
    float* const __restrict__ res35, float* const __restrict__ res36,
    float* const __restrict__ res37, float* const __restrict__ res38,
    float* const __restrict__ res39, float* const __restrict__ res40,
    float* const __restrict__ res41, float* const __restrict__ res42,
    float* const __restrict__ res43, float* const __restrict__ res44,
    float* const __restrict__ res45, float* const __restrict__ res46,
    float* const __restrict__ res47, float* const __restrict__ res48,
    float* const __restrict__ res49, float* const __restrict__ res50,
    float* const __restrict__ res51, float* const __restrict__ res52,
    float* const __restrict__ res53, float* const __restrict__ res54,
    float* const __restrict__ res55, float* const __restrict__ res56,
    float* const __restrict__ res57) {
  // Total ops: 53

  // Intermediate terms (5)
  const float _tmp0 = M_LN2;
  const float _tmp1 = M_1_PI;
  const float _tmp2 = M_SQRT2;
  const float _tmp3 = sqrtf(x);
  const float _tmp4 = x + y;

  // Output terms (58)
  if (res0 != nullptr) {
    *res0 = 0;
  }

  if (res1 != nullptr) {
    *res1 = 1;
  }

  if (res2 != nullptr) {
    *res2 = 1.0F / 2.0F;
  }

  if (res3 != nullptr) {
    *res3 = 1.6F;
  }

  if (res4 != nullptr) {
    *res4 = 11.0F / 52.0F;
  }

  if (res5 != nullptr) {
    *res5 = M_E;
  }

  if (res6 != nullptr) {
    *res6 = 1.0F / _tmp0;
  }

  if (res7 != nullptr) {
    *res7 = _tmp0;
  }

  if (res8 != nullptr) {
    *res8 = M_LN10;
  }

  if (res9 != nullptr) {
    *res9 = M_PI;
  }

  if (res10 != nullptr) {
    *res10 = M_PI_2;
  }

  if (res11 != nullptr) {
    *res11 = M_PI_4;
  }

  if (res12 != nullptr) {
    *res12 = _tmp1;
  }

  if (res13 != nullptr) {
    *res13 = 2 * _tmp1;
  }

  if (res14 != nullptr) {
    *res14 = M_2_SQRTPI;
  }

  if (res15 != nullptr) {
    *res15 = _tmp2;
  }

  if (res16 != nullptr) {
    *res16 = (1.0F / 2.0F) * _tmp2;
  }

  if (res17 != nullptr) {
    *res17 = fabsf(x);
  }

  if (res18 != nullptr) {
    *res18 = sinf(x);
  }

  if (res19 != nullptr) {
    *res19 = cosf(x);
  }

  if (res20 != nullptr) {
    *res20 = tanf(x);
  }

  if (res21 != nullptr) {
    *res21 = asinf(x);
  }

  if (res22 != nullptr) {
    *res22 = acosf(x);
  }

  if (res23 != nullptr) {
    *res23 = atanf(x);
  }

  if (res24 != nullptr) {
    *res24 = expf(x);
  }

  if (res25 != nullptr) {
    *res25 = logf(x);
  }

  if (res26 != nullptr) {
    *res26 = sinhf(x);
  }

  if (res27 != nullptr) {
    *res27 = coshf(x);
  }

  if (res28 != nullptr) {
    *res28 = tanhf(x);
  }

  if (res29 != nullptr) {
    *res29 = floorf(x);
  }

  if (res30 != nullptr) {
    *res30 = ceilf(x);
  }

  if (res31 != nullptr) {
    *res31 = _tmp3;
  }

  if (res32 != nullptr) {
    *res32 = asinhf(x);
  }

  if (res33 != nullptr) {
    *res33 = acoshf(x);
  }

  if (res34 != nullptr) {
    *res34 = atanhf(x);
  }

  if (res35 != nullptr) {
    *res35 = fmodf(1.0F * x, 5.5F);
  }

  if (res36 != nullptr) {
    *res36 = x + 1;
  }

  if (res37 != nullptr) {
    *res37 = 2 * x;
  }

  if (res38 != nullptr) {
    *res38 = powf(x, 2);
  }

  if (res39 != nullptr) {
    *res39 = powf(x, 3);
  }

  if (res40 != nullptr) {
    *res40 = powf(x, 4);
  }

  if (res41 != nullptr) {
    *res41 = powf(x, 5);
  }

  if (res42 != nullptr) {
    *res42 = _tmp3;
  }

  if (res43 != nullptr) {
    *res43 = powf(x, 3.0F / 2.0F);
  }

  if (res44 != nullptr) {
    *res44 = 0.5F * ((((x) >= 0) - ((x) < 0)) + 1);
  }

  if (res45 != nullptr) {
    *res45 = erfcf(x);
  }

  if (res46 != nullptr) {
    *res46 = lgammaf(x);
  }

  if (res47 != nullptr) {
    *res47 = erff(x);
  }

  if (res48 != nullptr) {
    *res48 = tgammaf(x);
  }

  if (res49 != nullptr) {
    *res49 = atan2f(x, y);
  }

  if (res50 != nullptr) {
    *res50 = fmaxf(x, y);
  }

  if (res51 != nullptr) {
    *res51 = fminf(x, y);
  }

  if (res52 != nullptr) {
    *res52 = fmodf(x, y);
  }

  if (res53 != nullptr) {
    *res53 = _tmp4;
  }

  if (res54 != nullptr) {
    *res54 = x * y;
  }

  if (res55 != nullptr) {
    *res55 = powf(x, y);
  }

  if (res56 != nullptr) {
    *res56 = powf(_tmp4, 2);
  }

  if (res57 != nullptr) {
    *res57 = powf(_tmp4, 3);
  }
}

}  // namespace sym
